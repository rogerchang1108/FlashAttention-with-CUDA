
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <float.h>
#include <sys/time.h>

void input(char *input_filename);
void output(char *output_filename);
__global__ void flash_attention_kernel(const float4* q, const float4* k, const float4* v, const int N, const int d,
                    const int tc, const int tr, const int bc, const int br, const float softmax_scale,
                    float *o);

int B, N, d;
float *Q, *K, *V, *O;

int main(int argc, char *argv[]) {
    input(argv[1]);

    const int br = 128, bc = (d == 64) ? 32 : 64;
    const int tr = N / br, tc = N / bc;
    const float softmax_scale = rsqrtf(d);

    const int vec_d = d / 4;
    const int sram_size = (2 * bc * vec_d * sizeof(float4)) + (bc * br * sizeof(float));

    float4 *h_q = (float4*)malloc(B * N * vec_d * sizeof(float4));
    float4 *h_k = (float4*)malloc(B * N * vec_d * sizeof(float4));
    float4 *h_v = (float4*)malloc(B * N * vec_d * sizeof(float4));

    for (int i = 0; i < B * N * vec_d; i++) {
        int base = i << 2;
        h_q[i] = make_float4(Q[base], Q[base+1], Q[base+2], Q[base+3]);
        h_k[i] = make_float4(K[base], K[base+1], K[base+2], K[base+3]);
        h_v[i] = make_float4(V[base], V[base+1], V[base+2], V[base+3]);
    }

    float4 *d_q, *d_k, *d_v;
    float *d_o;
    
    hipMalloc(&d_q, B * N * vec_d * sizeof(float4));
    hipMalloc(&d_k, B * N * vec_d * sizeof(float4));
    hipMalloc(&d_v, B * N * vec_d * sizeof(float4));
    hipMalloc(&d_o, B * N * d * sizeof(float));

    hipMemcpy(d_q, h_q, B * N * vec_d * sizeof(float4), hipMemcpyHostToDevice);
    hipMemcpy(d_k, h_k, B * N * vec_d * sizeof(float4), hipMemcpyHostToDevice);
    hipMemcpy(d_v, h_v, B * N * vec_d * sizeof(float4), hipMemcpyHostToDevice);
    hipMemset(d_o, 0x00, B * N * d * sizeof(float));

    dim3 grid_dim(B, tr);
    dim3 block_dim(br);
    
    flash_attention_kernel<<<grid_dim, block_dim, sram_size>>>(
        d_q, d_k, d_v, N, d, tc, tr, bc, br, softmax_scale, d_o
    );

    hipMemcpy(O, d_o, B * N * d * sizeof(float), hipMemcpyDeviceToHost);

    free(h_q);
    free(h_k);
    free(h_v);

    hipFree(d_q);
    hipFree(d_k);
    hipFree(d_v);
    hipFree(d_o);


    output(argv[2]);
    return 0;
}

void input(char *input_filename) {
    FILE *file = fopen(input_filename, "rb");

    fread(&B, sizeof(int), 1, file);
    fread(&N, sizeof(int), 1, file);
    fread(&d, sizeof(int), 1, file);

    Q = (float *)malloc(B * N * d * sizeof(float));
    K = (float *)malloc(B * N * d * sizeof(float));
    V = (float *)malloc(B * N * d * sizeof(float));
    O = (float *)malloc(B * N * d * sizeof(float));

    for (int i = 0; i < B; i++) {
        fread(Q + (i * N * d), sizeof(float), N * d, file);
        fread(K + (i * N * d), sizeof(float), N * d, file);
        fread(V + (i * N * d), sizeof(float), N * d, file);
    }

    fclose(file);
}

void output(char *output_filename) {
    FILE *file = fopen(output_filename, "wb");

    fwrite(O, sizeof(float), B * N * d, file);

    free(Q);
    free(K);
    free(V);
    free(O);

    fclose(file);
}

__global__ void flash_attention_kernel(const float4* q, const float4* k, const float4* v, const int N, const int d,
                    const int tc, const int tr, const int bc, const int br, const float softmax_scale,
                    float *o) {
    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x;

    const int vec_d = d / 4;
    int qkv_offset = (bx * N * vec_d);

    extern __shared__ float4 sram[];
    int tile_size_r = br * vec_d;
    int tile_size_c = bc * vec_d;
    float4* kj = sram;
    float4* vj = &sram[tile_size_c];
    float* sij = (float*)&sram[tile_size_c * 2];
    
    float4 qi[16]; 

    int i = by;
    // Load Q
    for (int x = 0; x < vec_d; ++x) {
        qi[x] = q[qkv_offset + (tile_size_r * i) + (tx * vec_d) + x];
    }

    float mi_old = 0.0f;
    float li_old = 0.0f;
    
    int step = br / vec_d;
    for (int j = 0; j < tc; ++j) {
        for (int x = 0; x < bc; x += step) {
            kj[(x * vec_d) + tx] = k[qkv_offset + (tile_size_c * j) + (x * vec_d) + tx];
            vj[(x * vec_d) + tx] = v[qkv_offset + (tile_size_c * j) + (x * vec_d) + tx];
        }
        __syncthreads();

        // QKDotAndScalar(sij) && RowMax(mij)
        float mij = FLT_MIN;
        
        for (int y = 0; y < bc; ++y) {
            float sum = 0.0f;
            for (int x = 0; x < vec_d; ++x) {
                float4 q4 = qi[x];
                float4 k4 = kj[(y * vec_d) + x];
                sum += q4.x * k4.x + q4.y * k4.y + q4.z * k4.z + q4.w * k4.w;
            }
            sum *= softmax_scale;
            sij[(y * br) + tx] = sum;
            mij = fmaxf(mij, sum);
        }
        // MinusMaxAndExp(pij->sij) && RowSum(lij)
        float lij = 0.0f;
        for (int y = 0; y < bc; ++y) {
            sij[(y * br) + tx] = __expf(sij[(y * br) + tx] - mij);
            lij += sij[(y * br) + tx];
        }
        // UpdateMiLiOi
        // Compute new m and l
        float mi_new = fmaxf(mi_old, mij);
        float exp_old = __expf(mi_old - mi_new);
        float exp_new = __expf(mij - mi_new);
        float li_new = (exp_old * li_old) + (exp_new * lij);

        float scale = __fdividef(1.0f, li_new);
        float old_scale = li_old * exp_old * scale;
        float exp_scale = exp_new * scale;

        for (int x = 0; x < vec_d; ++x) {
            float4 pv = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
            for (int y = 0; y < bc; ++y) {
                float s = sij[(y * br) + tx];
                float4 v4 = vj[y * vec_d + x];
                pv.x += s * v4.x;
                pv.y += s * v4.y;
                pv.z += s * v4.z;
                pv.w += s * v4.w;
            }
            
            int base_idx = (qkv_offset + (tile_size_r * i) + (tx * vec_d) + x) << 2;
            
            o[base_idx] = old_scale * o[base_idx] + exp_scale * pv.x;
            o[base_idx + 1] = old_scale * o[base_idx + 1] + exp_scale * pv.y;
            o[base_idx + 2] = old_scale * o[base_idx + 2] + exp_scale * pv.z;
            o[base_idx + 3] = old_scale * o[base_idx + 3] + exp_scale * pv.w;
        }

        mi_old = mi_new;
        li_old = li_new;
        __syncthreads();
    }
}