
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <float.h>
#include <sys/time.h>
// Remove l, m Array

void input(char *input_filename);
void output(char *output_filename);
__global__ void flash_attention_kernel(const float4* q, const float4* k, const float4* v, const int N, const int d,
                    const int tc, const int tr, const int bc, const int br, const float softmax_scale,
                    float *o);

double getTimeStamp() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (double) tv.tv_usec/1000000 + tv.tv_sec;
}

int B, N, d;
float *Q, *K, *V, *O;

int main(int argc, char *argv[]) {
    double start, end, input_end, output_start;
    start = getTimeStamp();

    input(argv[1]);

    input_end = getTimeStamp();

    const int br = 128, bc = (d == 64) ? 16 : 32;
    const int tr = N / br, tc = N / bc;
    const float softmax_scale = rsqrtf(d);

    const int vec_d = d / 4;
    const int sram_size = (2 * bc * vec_d * sizeof(float4)) + (br * vec_d * sizeof(float4)) + (bc * br * sizeof(float));
    int max_sram_size;
    hipDeviceGetAttribute(&max_sram_size, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
    printf("Max shared memory: %d, requested shared memory: %d \n", max_sram_size, sram_size);

    float4 *h_q = (float4*)malloc(B * N * vec_d * sizeof(float4));
    float4 *h_k = (float4*)malloc(B * N * vec_d * sizeof(float4));
    float4 *h_v = (float4*)malloc(B * N * vec_d * sizeof(float4));

    for (int i = 0; i < B * N * vec_d; i++) {
        int base = i << 2;
        h_q[i] = make_float4(Q[base], Q[base+1], Q[base+2], Q[base+3]);
        h_k[i] = make_float4(K[base], K[base+1], K[base+2], K[base+3]);
        h_v[i] = make_float4(V[base], V[base+1], V[base+2], V[base+3]);
    }

    float4 *d_q, *d_k, *d_v;
    float *d_o;
    
    hipMalloc(&d_q, B * N * vec_d * sizeof(float4));
    hipMalloc(&d_k, B * N * vec_d * sizeof(float4));
    hipMalloc(&d_v, B * N * vec_d * sizeof(float4));
    hipMalloc(&d_o, B * N * d * sizeof(float));

    hipMemcpy(d_q, h_q, B * N * vec_d * sizeof(float4), hipMemcpyHostToDevice);
    hipMemcpy(d_k, h_k, B * N * vec_d * sizeof(float4), hipMemcpyHostToDevice);
    hipMemcpy(d_v, h_v, B * N * vec_d * sizeof(float4), hipMemcpyHostToDevice);
    hipMemset(d_o, 0x00, B * N * d * sizeof(float));

    dim3 grid_dim(B, tr);
    dim3 block_dim(br);
    
    flash_attention_kernel<<<grid_dim, block_dim, sram_size>>>(
        d_q, d_k, d_v, N, d, tc, tr, bc, br, softmax_scale, d_o
    );

    hipMemcpy(O, d_o, B * N * d * sizeof(float), hipMemcpyDeviceToHost);

    free(h_q);
    free(h_k);
    free(h_v);

    hipFree(d_q);
    hipFree(d_k);
    hipFree(d_v);
    hipFree(d_o);

    output_start = getTimeStamp();

    output(argv[2]);

    end = getTimeStamp();
    printf("(B, N, d): (%d, %d, %d)\n", B, N, d);
    printf("Total Time: %.3f seconds\n", end - start);
    printf("Input Time: %.3f seconds\n", input_end - start);
    printf("Output Time: %.3f seconds\n", end - output_start);
    return 0;
}

void input(char *input_filename) {
    FILE *file = fopen(input_filename, "rb");

    fread(&B, sizeof(int), 1, file);
    fread(&N, sizeof(int), 1, file);
    fread(&d, sizeof(int), 1, file);

    Q = (float *)malloc(B * N * d * sizeof(float));
    K = (float *)malloc(B * N * d * sizeof(float));
    V = (float *)malloc(B * N * d * sizeof(float));
    O = (float *)malloc(B * N * d * sizeof(float));

    for (int i = 0; i < B; i++) {
        fread(Q + (i * N * d), sizeof(float), N * d, file);
        fread(K + (i * N * d), sizeof(float), N * d, file);
        fread(V + (i * N * d), sizeof(float), N * d, file);
    }

    fclose(file);
}

void output(char *output_filename) {
    FILE *file = fopen(output_filename, "wb");

    fwrite(O, sizeof(float), B * N * d, file);

    free(Q);
    free(K);
    free(V);
    free(O);

    fclose(file);
}

__global__ void flash_attention_kernel(const float4* q, const float4* k, const float4* v, const int N, const int d,
                    const int tc, const int tr, const int bc, const int br, const float softmax_scale,
                    float *o) {
    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x;

    const int vec_d = d / 4;
    int qkv_offset = (bx * N * vec_d);

    extern __shared__ float4 sram[];
    int tile_size_r = br * vec_d;
    int tile_size_c = bc * vec_d;
    float4* qi = sram;
    float4* kj = &sram[tile_size_r];
    float4* vj = &sram[tile_size_r + tile_size_c];
    float* sij = (float*)&sram[tile_size_r + tile_size_c * 2];

    int i = by;
    // Load Q
    int step = br / vec_d;
    for (int x = 0; x < br; x += step) {
        qi[(x * vec_d) + tx] = q[qkv_offset + (tile_size_r * i) + (x * vec_d) + tx];
    }

    float mi_old = 0.0f;
    float li_old = 0.0f;
    
    for (int j = 0; j < tc; ++j) {
        for (int x = 0; x < bc; x += step) {
            kj[(x * vec_d) + tx] = k[qkv_offset + (tile_size_c * j) + (x * vec_d) + tx];
            vj[(x * vec_d) + tx] = v[qkv_offset + (tile_size_c * j) + (x * vec_d) + tx];
        }
        __syncthreads();

        // QKDotAndScalar(sij) && RowMax(mij)
        float mij = FLT_MIN;
        
        for (int y = 0; y < bc; ++y) {
            float sum = 0.0f;
            for (int x = 0; x < vec_d; ++x) {
                float4 q4 = qi[(tx * vec_d) + x];
                float4 k4 = kj[(y * vec_d) + x];
                sum += q4.x * k4.x + q4.y * k4.y + q4.z * k4.z + q4.w * k4.w;
            }
            sum *= softmax_scale;
            sij[(tx * bc) + y] = sum;
            mij = fmaxf(mij, sum);
        }
        // MinusMaxAndExp(pij->sij) && RowSum(lij)
        float lij = 0.0f;
        for (int y = 0; y < bc; ++y) {
            sij[(tx * bc) + y] = __expf(sij[(tx * bc) + y] - mij);
            lij += sij[(tx * bc) + y];
        }
        // UpdateMiLiOi
        // Compute new m and l
        float mi_new = fmaxf(mi_old, mij);
        float exp_old = __expf(mi_old - mi_new);
        float exp_new = __expf(mij - mi_new);
        float li_new = (exp_old * li_old) + (exp_new * lij);

        float scale = __fdividef(1.0f, li_new);
        float old_scale = li_old * exp_old * scale;
        float exp_scale = exp_new * scale;

        for (int x = 0; x < vec_d; ++x) {
            float4 pv = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
            for (int y = 0; y < bc; ++y) {
                float s = sij[(tx * bc) + y];
                float4 v4 = vj[y * vec_d + x];
                pv.x += s * v4.x;
                pv.y += s * v4.y;
                pv.z += s * v4.z;
                pv.w += s * v4.w;
            }
            
            int base_idx = (qkv_offset + (tile_size_r * i) + (tx * vec_d) + x) << 2;
            
            o[base_idx] = old_scale * o[base_idx] + exp_scale * pv.x;
            o[base_idx + 1] = old_scale * o[base_idx + 1] + exp_scale * pv.y;
            o[base_idx + 2] = old_scale * o[base_idx + 2] + exp_scale * pv.z;
            o[base_idx + 3] = old_scale * o[base_idx + 3] + exp_scale * pv.w;
        }

        mi_old = mi_new;
        li_old = li_new;
        __syncthreads();
    }
}